#include "hip/hip_runtime.h"
#include <fmt/core.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void cuda_hello() { printf("Hello World from GPU!\n"); }

int main() {
  cuda_hello<<<1, 1>>>();
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

  fmt::print("fmt test \n");
  return 0;
}